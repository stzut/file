#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

static inline
int getBlocks(long working_set_size, int threadsPerBlock) {
  // TODO: implement and use this interface if necessary
    int Block_num = (working_set_size + threadsPerBlock -1) / threadsPerBlock;
    return Block_num;  
}

void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
  // TODO: implement and use this interface if necessary 
  // we will not use it 
}

void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
  // TODO: implement and use this interface if necessary 
    hipFree(xarray);
    hipFree(yarray);
    hipFree(resultarray); 
}

void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 512; // change this if necessary

    float *device_x;
    float *device_y;
    float *device_result;

    //
    // TODO: do we need to allocate device memory buffers on the GPU here?
    //
    hipMallocManaged(&device_x, total_elems*sizeof(float));
    hipMallocManaged(&device_y, total_elems*sizeof(float));
    hipMallocManaged(&device_result, total_elems*sizeof(float));
    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

  
    //
    // TODO: do we need copy here?
    // No! because unified memory is acessible by both CPU and GPU
     
    //
    // TODO: insert time here to begin timing only the kernel
    //
    double startGPUTime = CycleTimer::currentSeconds();
    // compute number of blocks and threads per block
    int Block_num = getBlocks(total_elems, threadsPerBlock);
    // run saxpy_kernel on the GPU
    saxpy_kernel<<<Block_num, threadsPerBlock>>>(total_elems, alpha, device_x, device_y, device_result);
    //
    // TODO: insert timer here to time only the kernel.  Since the
    // kernel will run asynchronously with the calling CPU thread, you
    // need to call hipDeviceSynchronize() before your timer to
    // ensure the kernel running on the GPU has completed.  (Otherwise
    // you will incorrectly observe that almost no time elapses!)
    //
    hipDeviceSynchronize();
    double endGPUTime = CycleTimer::currentSeconds();
    double timeKernel = endGPUTime - startGPUTime;
    
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
    
    //
    // TODO: copy result from GPU using hipMemcpy
    //
    double start_dev2host_Time = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, total_elems * sizeof(float), hipMemcpyDeviceToHost);
    
    double end_dev2host_Time = CycleTimer::currentSeconds();
    double timedev2Host = end_dev2host_Time - start_dev2host_Time;

    // What would be copy time when we use UVM?
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    totalTimeAvg   += overallDuration;
    timeKernelAvg  += timeKernel;
    timeCopyD2HAvg += timedev2Host;

    //
    // TODO free device memory if you allocate some device memory earlier in this function.
    //
    freeArrays(device_x, device_y, device_result);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
